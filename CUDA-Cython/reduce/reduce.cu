#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#include "reduce.hpp"

const int block_size = 128;


void __global__ reduce_kernel(const int N, float *v_dev, float *res_dev)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;

    __shared__ float s_y[128];
    s_y[tid] = (n < N) ? v_dev[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        res_dev[bid] = s_y[0];
    }
}

void reduce(int N, float *input, float *output)
{
    float *v_dev; // define device vector pointer
    const int M = sizeof(float) * N;
    CHECK(hipMalloc(&v_dev, M));
    CHECK(hipMemcpy(v_dev, input, M, hipMemcpyHostToDevice));

    int grid_size = (N + block_size - 1) / block_size;
    // allocate memory for result vector in device
    const int res_mem = sizeof(float) * grid_size;
    float *res_dev;
    CHECK(hipMalloc(&res_dev, res_mem));

    // kernel
    reduce_kernel<<<grid_size, block_size>>>(N, v_dev, res_dev);

    // allocate memory for result vector in host
    float *res_h = (float *) malloc(res_mem);
    CHECK(hipMemcpy(res_h, res_dev, res_mem, hipMemcpyDeviceToHost));

    float _temp = 0;
    for (int i = 0; i < grid_size; ++i)
    {
        _temp += res_h[i];
    }
    *output = _temp;

    CHECK(hipFree(v_dev));
    CHECK(hipFree(res_dev));
    free(res_h);
}